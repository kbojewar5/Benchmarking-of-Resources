// Memorycuda.cu
// GPU Benchmarking
//
// Created by Ronakkumar Makadiya, Kaustubh Bojewar, Sourabh //CHougale 
// Created on 09/09/2104
// Copyright (c) 2014 Ronakkumar Makadiya. All rights reserved.
//


#include "hip/hip_runtime.h"

#include <string.h>
#include <stdlib.h>
#include <stdio.h>

void calclulate(float timeTaken, int no_of_threads){
	float data = 0.001;
	printf("\n%f GB/sec", (data / (timeTaken / 1024.0))*no_of_threads);

}


__global__ void readwritebyte(char *str, int *size){

	char* a_to_z = "ABCDEFGHIJKLMNOPQRSTUVWXYZ";
	char str2 = a_to_z[0];
	char *s = &str2;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	for (int i = 0; i < 1024*1024; i++){
		memcpy(&str[index], s, sizeof(char));
	}
	
	//free(s);
}



void startkernal(int threads,int blocks,int* size)
{

	//cudaError_t cudaStatus;
	hipEvent_t start, stop;
	char *str_d;
	int *size_d;
	float time;

	hipMalloc((void**)&str_d, *size * sizeof(char));
	hipMalloc((void**)&size_d, sizeof(int));
	hipMemcpy(size_d,size, sizeof(int), hipMemcpyHostToDevice);


		
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	readwritebyte <<<blocks, threads >>>(str_d,size_d);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\nTime for read and write one Byte kernel: %f ms", time);

	calclulate(time, blocks*threads);
}	


int main()
{

	int num_of_blocks=1024;
	int num_of_threads_block=1024;
	int mem_size=1024*1024*1024;

	startkernal(num_of_threads_block,num_of_blocks,&mem_size);

	return 0;

}
