//
// flops.cu
// GPU Benchmarking
//
// Created by Ronakkumar Makadiya, Kaustubh Bojewar, Sourabh //CHougale 
// Created on 09/09/2104
// Copyright (c) 2014 Ronakkumar Makadiya. All rights reserved.
//

#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
using namespace std;
#define SIZE 10000000


__global__ void emptyLoopTime(int n)
{

	long int i=0;
	int a=0;
	for(i=0;i<n;i++)
	{
	}

}

__global__ void flopsCUDA(long int* total,int n)
{
	//clock_t t1,t2,total_time=0;
	long int i=0;
	int a=0;
	
	for(i=0;i<n;i++)
	{
		a=a+0.5;
	}

}


void calculateFlops() 
{

	long int total=0;
	long int *d_total;
	double time=0;
	double *d_time;

	//cudaError_t cudaStatus;
	hipMalloc(&d_total, sizeof(long int));
	hipMalloc(&d_time, sizeof(double));

	hipMemcpy(d_total,&total, sizeof(long int),hipMemcpyHostToDevice);
	hipMemcpy(d_time,&time, sizeof(double),hipMemcpyHostToDevice);
//------------------------------------------------------------------------
	hipEvent_t empty_start, empty_stop;
	hipEventCreate(&empty_start);
	hipEventCreate(&empty_stop);

// Start record
	
	hipEventRecord(empty_start, 0);
	
	emptyLoopTime<<<1,1>>>(SIZE);

	hipEventRecord(empty_stop, 0);

	hipEventSynchronize(empty_start); //optional
	hipEventSynchronize(empty_stop);

	float emptyloop;
	hipEventElapsedTime(&emptyloop,empty_start,empty_stop);

	hipEventDestroy(empty_start);
	hipEventDestroy(empty_stop); 
	//cout << "Empty Loop time:"<<emptyloop<<endl;

//------------------------------------------------------------------
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
//	cudaStatus = cudaDeviceSynchronize();
   	 

	hipEventRecord(start, 0);
	flopsCUDA<<<1,1>>>(d_total,SIZE);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start); //optional
	hipEventSynchronize(stop);

	
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time! Clean up:
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

//-----------------------------------------------------------------------
	//cout << "Time elapsed:"<<elapsedTime<<endl;

	hipMemcpy(&total,d_total,sizeof(long int),hipMemcpyDeviceToHost);
	hipMemcpy(&time,d_time,sizeof(double),hipMemcpyDeviceToHost);

	long double flops=SIZE/(emptyloop-elapsedTime);

	cout<<"\nFLOPS:"<<flops<<endl;

	double gflops=flops/1000000000;

	cout<< "\n The GFLOPS:"<<gflops;
	//cout<<"\nThe answer is "<<total<<endl;
//cout<<"The answer is "<<b<<endl;

	hipFree(d_total);
	hipFree(d_time);
	

}


int main(){

	cout <<"\n\n---------------------GFLOPS CUDA Benchmarking------------------------------------\n\n";
	calculateFlops();


	//cout <<"\n\n---------------------GIOPS CUDA Benchmarking------------------------------------\n\n";	
	//calculateIops();

	
	
	return 0;
}

